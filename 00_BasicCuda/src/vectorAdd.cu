#include <iostream>
#include <hip/hip_runtime.h>


// kernel for vector Addition
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void runVectorAdd(int N){
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    // Initialize host arrays
    for (int i = 0; i< N; i++){
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A , size);
    hipMalloc((void**)&d_B , size);
    hipMalloc((void**)&d_C , size);

    // copy data from host to device
    hipMemcpy( d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy( d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_B,d_C,N);

    // Copy results from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; ++i) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << "!" << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    std::cout << "Test PASSED" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

}

int main(){
    int N = 1 << 20; 
    printf("Vector addition example with %d elements\n", N);
    runVectorAdd(N);
    return 0;
}